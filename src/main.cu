#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <string>
#include <vector>
#include <random>
#include <chrono>
#include <stdexcept>
#include <functional>
#include <float.h>

#include "sum_search.cuh"
#include "scan.cuh"

using namespace std;
typedef chrono::high_resolution_clock clck;

// This will be slower than using hiprand, but we want to check accuracy/speed
// against the naiive linear CPU implementation.
template<typename T>
vector<T> get_random_array(long N, T (*rand_T)()) {
  vector<T> vec{};
  vec.reserve(N);
  for (long i = 0; i < N; ++i) vec.emplace_back(rand_T());
  return vec;
}

enum ArrayType {
  I32,
  I64,
  F32,
  F64
};

struct Parameters {
  long seed = 123456789;
  long num_tests = 100;
  long numel = 10000;
  long double max = INT_MAX;
  int device = 0;
  ArrayType vtype = I32;
};

// Parse command line arguments
// Throws runtime_error for invalid input
Parameters parse_args(int argc, char** argv) {
  Parameters params{};
  int iarg = 1;
  while (iarg < argc) {
    if (strcmp(argv[iarg], "-s")==0 || strcmp(argv[iarg], "--seed")==0) {
      if (++iarg >= argc) throw runtime_error("Missing argument for seed");
      params.seed = strtol(argv[iarg++], nullptr, 10);
    } else if (strcmp(argv[iarg], "-d")==0 || strcmp(argv[iarg], "--device")==0) {
      if (++iarg >= argc) throw runtime_error("Missing argument for device");
      params.device = static_cast<int>(strtol(argv[iarg++], nullptr, 10));
    } else if (strcmp(argv[iarg], "-n")==0 || strcmp(argv[iarg], "--num-tests")==0) {
      if (++iarg >= argc) throw runtime_error("Missing argument for number of tests");
      params.num_tests = strtol(argv[iarg++], nullptr, 10);
    } else if (strcmp(argv[iarg], "-N")==0 || strcmp(argv[iarg], "--num-elements")==0) {
      if (++iarg >= argc) throw runtime_error("Missing argument for number of elements");
      params.numel = strtol(argv[iarg++], nullptr, 10);
    } else if (strcmp(argv[iarg], "-t")==0 || strcmp(argv[iarg], "--type")==0) {
      if (++iarg >= argc) throw runtime_error("Missing argument for type");
      if (strcmp(argv[iarg], "i32")==0 || strcmp(argv[iarg], "I32")==0) {
        params.vtype = ArrayType::I32;
        params.max = INT_MAX;
      } else if (strcmp(argv[iarg], "i64")==0 || strcmp(argv[iarg], "I64")==0) {
        params.vtype = ArrayType::I64;
        params.max = LONG_MAX;
      } else if (strcmp(argv[iarg], "f32")==0 || strcmp(argv[iarg], "F32")==0) {
        params.vtype = ArrayType::F32;
        params.max = FLT_MAX;
      } else if (strcmp(argv[iarg], "f64")==0 || strcmp(argv[iarg], "F64")==0) {
        params.vtype = ArrayType::F64;
        params.max = DBL_MAX;
      } else throw runtime_error("Unknown data type");
      ++iarg;
    } else throw runtime_error("Unknown argument '" + string(argv[iarg]) + "'");
  }
  if (params.vtype == ArrayType::I32 || params.vtype == ArrayType::I64)
    params.max /= params.numel;
  return params;
}


template<typename DIST>
void test_partial_scan(const Parameters &params, DIST &dist, mt19937 &engine) {
  typedef typename DIST::result_type T;
  vector<T> vec(params.numel);
  T* vec_in_d; hipMalloc((void**)&vec_in_d, sizeof(T)*params.numel);
  T* vec_out_d; hipMalloc((void**)&vec_in_d, sizeof(T)*params.numel);

  static const int block_size = 512;

  clck::duration time_tot{0};
  clck::time_point time_start;
  for (long i = 0; i < params.num_tests; ++i) {
    for (long n = 0; n < params.numel; ++n) vec[n] = dist(engine);
    hipMemcpy(vec_in_d, vec.data(), sizeof(T)*params.numel, hipMemcpyHostToDevice);
    time_start = clck::now();

    scan_up_sweep<<<(params.numel+block_size-1)/block_size, block_size, sizeof(T)*block_size*2>>>(vec_in_d, vec_out_d);

    auto time_end = clck::now();
    time_tot += time_end - time_start;
  }
  printf("Partial scan, search on GPU\t%10g ns\n", static_cast<double>(time_tot.count()) / params.num_tests);
}

void measure_partial_scan(const Parameters &params) {
  hipSetDevice(params.device);

  mt19937 engine{};
  engine.seed(params.seed);

  switch (params.vtype) {
    case ArrayType::I32:
      {
        uniform_int_distribution<int> dist{0, static_cast<int>(params.max)};
        test_partial_scan(params, dist, engine);
        break;
      }
    case ArrayType::I64:
      {
        uniform_int_distribution<long> dist{0, static_cast<long>(params.max)};
        test_partial_scan(params, dist, engine);
        break;
      }
    case ArrayType::F32:
      {
        uniform_real_distribution<float> dist{0, static_cast<float>(params.max)};
        test_partial_scan(params, dist, engine);
        break;
      }
    case ArrayType::F64:
      {
        uniform_real_distribution<double> dist{0, static_cast<double>(params.max)};
        test_partial_scan(params, dist, engine);
        break;
      }
  }
}

int main(int argc, char** argv) {
  // Parse flags
  Parameters params;
  try {
    params = parse_args(argc, argv);
  } catch (runtime_error e) {
    printf("Error parsing command line flags:\n%s\n", e.what());
    return 1;
  }

  measure_partial_scan(params);


  return 0;
}
